
#include <hip/hip_runtime.h>
#include <stdio.h>

static const int BLOCKS = 1024;
static const int THREADS = 1024;

__global__ void vector_sum(unsigned int *a,unsigned int *b,unsigned int *result){
	unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid<BLOCKS*THREADS){
		result[tid]=a[tid]+b[tid];
	}

}

int main(int argc, char **argv)
{
	int N = BLOCKS*THREADS;

	unsigned int *a = (unsigned int*)malloc(sizeof(unsigned int)*N);
	unsigned int *b = (unsigned int*)malloc(sizeof(unsigned int)*N);
	unsigned int *result = (unsigned int*)malloc(sizeof(unsigned int)*N);

	unsigned int *d_a, *d_b, *d_result;

	hipMalloc( (void**)&d_a, N * sizeof(unsigned int));
	hipMalloc( (void**)&d_b, N * sizeof(unsigned int));
	hipMalloc( (void**)&d_result, N * sizeof(unsigned int));

	for (int i=0; i<N; i++){
		a[i] = i;
		b[i] = i*i;
	}

	hipMemcpy( d_a, a, N * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( d_b, b, N * sizeof(int),hipMemcpyHostToDevice);

	vector_sum<<<BLOCKS,THREADS>>>(d_a,d_b,d_result);
	hipMemcpy(result, d_result, N * sizeof(unsigned int),hipMemcpyDeviceToHost);
	printf("Device done\n");
	double total = 0;
	for (int i=0; i<N; i++) {
		total+=result[i];
	}
	printf( "%f\n",total);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_result );
	return 0;
}
