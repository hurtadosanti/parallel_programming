#include "hip/hip_runtime.h"
__global__ void index_print_kernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_idx = threadIdx.x / warpSize; 
    int lane_idx = threadIdx.x & (warpSize 1); 
    if ((lane_idx & (warpSize/2-1)) == 0){
         //thread, block, warp, lane 
        printf(" %5d\t%5d\t %2d\t%2d\n", idx, blockIdx.x, warp_idx, lane_idx); 
    }
} 
int main() { 
    int gridDim = 4, blockDim = 128;
    puts("thread, block, warp, lane");
    index_print_kernel<<< gridDim, blockDim >>>(); 
    hipDeviceSynchronize(); 
    return 0;
} 

//nvcc m64 o cuda_thread_block cuda_thread_block.cu (Kindle Location 1629). 


