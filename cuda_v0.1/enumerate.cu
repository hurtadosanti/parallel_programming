#include "hip/hip_runtime.h"
#include <stdio.h>

/**
 * Simple cuda program to show how to uniquely identify all the treads 
 */
__global__ void enumerate() {
    int count = blockIdx.x * blockDim.x + threadIdx.x;
    printf("(%d * %d + %d )=%d\n", blockIdx.x, blockDim.x, threadIdx.x, count);
}

int main(void) {
    enumerate << < 4, 2 >> > ();
    hipDeviceSynchronize();
    return 0;
}
