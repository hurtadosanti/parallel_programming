#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

__device__ bool is_prime(unsigned int &num) {
    if (num == 2) {
        return true;
    }
    if (num < 2 || num % 2 == 0) {
        return false;
    } else {
        for (int i = 2; i < num / 2; i++) {
            if (num % i == 0) {
                return false;
            }
        }
    }
    return true;
}


__global__ void calculate_primes(unsigned int *result) {
    unsigned int count = blockIdx.x * blockDim.x + threadIdx.x;
    bool prime = is_prime(count);
    if (prime) {
        result[count] = count;
    } else {
        result[count] = 0;
    }
}

/**
 * Calculate the sum of primes
 */
int main(void) {
    clock_t start,end;
    double sum = 0;
    int blocks = 10;
    int threads = 10;
    unsigned int *result;
    unsigned int *primes = (unsigned int *) malloc(sizeof(unsigned int) * blocks * threads);

    hipMalloc((void **) &result, blocks * threads * sizeof(int));
    start=clock();
    calculate_primes << < blocks, threads >> > (result);
    hipMemcpy(primes, result, blocks * threads * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    end=clock();
    printf("device done\n");

    for (int i = 0; i < blocks * threads; i++) {
        sum += primes[i];
    }
    printf("Total: %f in %d seconds \n", sum,(double) (end-start)/CLOCKS_PER_SEC);

    hipFree(result);
    free(primes);
    return 0;
}
