
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>

__device__
bool is_prime(unsigned int number) {
    if (number == 2) {
        return true;
    } else if (number < 2 || number % 2 == 0) {
        return false;
    } else {
        // A faster option is to use math sqr root
        for (int i = 2; i < number / 2; ++i) {
            if (number % i == 0) {
                return false;
            }
        }
    }
    return true;
}

__global__ 
void calculate_primes(unsigned int* result, int size){
    auto count = blockIdx.x * blockDim.x + threadIdx.x;
    if(count<size){
        if(is_prime(count)){
            result[count]=count;
        }else{
            result[count]=0;
        }
    }
}
int main(){
    std::vector<unsigned int> result;
    double sum=0;
    int size = 1024*128;
    unsigned int *d_result;
    auto h_result = new unsigned int[size];
    hipMalloc((void**)&d_result,size* sizeof(unsigned int));
    calculate_primes<<<(size+1023)/1024,1024>>>(d_result,size);
    hipMemcpy(h_result,d_result,size* sizeof(unsigned int),hipMemcpyDeviceToHost);
    for(auto i=0;i<size;i++){
        auto value = h_result[i];
        if(value>0){
            result.push_back(value);
            sum+=value;
        }
    }
    hipFree(d_result);
    delete [] h_result;
    printf("%f for %d\n",sum,size);
    return 0;
}