#include <iostream>
#include <random>
#include "Sorting.h"

int main() {
    auto s = Sorting();

    std::random_device rd;
    std::vector<int> a(10000);
    for (auto& d : a) {
        d = static_cast<int>(rd());
    }


    auto D = s.cuda_sort(a);

    float free_m,total_m,used_m;

    size_t free_t,total_t;

    hipMemGetInfo(&free_t,&total_t);

    free_m =(uint)free_t/1048576.0 ;

    total_m=(uint)total_t/1048576.0;

    used_m=total_m-free_m;

    printf ( "  mem free %d .... %f MB mem total %d....%f MB mem used %f MB\n",free_t,free_m,total_t,total_m,used_m);

    // print contents of D
    for(int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;

    // H and D are automatically deleted when the function returns
    return 0;
}
