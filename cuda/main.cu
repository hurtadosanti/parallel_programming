#include <iostream>


int main() {
    // H has storage for 4 integers
    thrust::host_vector<int> H(4);

    // initialize individual elements
    H[0] = 49;
    H[1] = 20;
    H[2] = 12;
    H[3] = 46;

    // H.size() returns the size of vector H
    std::cout << "H has size " << H.size() << std::endl;

    // print contents of H
    for(int i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << std::endl;

    // resize H
    H.resize(2);


    std::cout << "H now has size " << H.size() << std::endl;

    // Copy host_vector H to device_vector D
    thrust::device_vector<int> D = H;

    // elements of D can be modified
    D[0] = 99;
    D[1] = 88;

    thrust::sort(D.begin(),D.end());
    float free_m,total_m,used_m;

    size_t free_t,total_t;

    hipMemGetInfo(&free_t,&total_t);

    free_m =(uint)free_t/1048576.0 ;

    total_m=(uint)total_t/1048576.0;

    used_m=total_m-free_m;

    printf ( "  mem free %d .... %f MB mem total %d....%f MB mem used %f MB\n",free_t,free_m,total_t,total_m,used_m);

    // print contents of D
    for(int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;

    // H and D are automatically deleted when the function returns
    return 0;
}
